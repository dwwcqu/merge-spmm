#include "hip/hip_runtime.h"
#ifndef GRB_BACKEND_APSPIE_KERNELS_SPMM_CU
#define GRB_BACKEND_APSPIE_KERNELS_SPMM_CU

#include <hip/hip_runtime.h>
#include <cstdio>
//#include <hip/hip_vector_types.h>

//#define TA     32
//#define TB     32
//#define NT     64

namespace graphblas
{
namespace backend
{
    typedef magma_index_t Index;

  // In paper "Design Principles for Sparse Matrix Multiplication"
  /*template<typename c, int TB>
  __global__ void spmmRowKernel2( const Index A_nrows, 
      const Index B_ncols, const Index A_ncols, const Index A_nvals,
      const Index* A_csrRowPtr, const Index* A_csrColInd, const c* A_csrVal, 
      const c* B_denseVal, c* C_denseVal )
  {
    float vals[TB];
    int   col_all[TB];
    float val_all[TB];

    int thread_id = blockDim.x*blockIdx.x+threadIdx.x; // global thrd idx
    int warp_id   = thread_id>>5;                      // global warp idx
    int lane_id   = thread_id & (32 - 1);
    int row       = warp_id;
    const c* B_offset = B_denseVal+lane_id+((blockIdx.y&1)<<5);
    int C_offset  = (row<<6)+lane_id+((blockIdx.y&1)<<5);

    //if( threadIdx.x==0 )
    //  printf("row:%d\n", row);

    if( row < A_nrows )
    {
      int row_start = __ldg(A_csrRowPtr+row);
      int row_end   = __ldg(A_csrRowPtr+row+1);

      int   col = -1;
      float val = 0.f;
      float sum = 0.f;
      int   jj  = row_start+lane_id;

      //TODO: add popc() and ballot to query which to shfl
      for( int jj_start=row_start; jj_start<row_end; jj_start+=32 )
      {
        //#pragma unroll
        //for( int ii=0; ii<TB; ii++ )
        //  vals[ii] = 0.f;
        if( jj<row_end )
        {
          col = __ldg(A_csrColInd+jj)<<6;
          val = __ldg(A_csrVal+jj);
        }
        else
        {
          col = 0;
          val = 0.f;
        }
        jj+=32;
        //if( warp_id==0 ) printf("tid:%d,col:%d,val:%f\n", threadIdx.x, col, val);
        for( int kk=0; kk<32; kk+=TB )
        {
          #pragma unroll
          for( int ii=0; ii<TB; ii++ )
          {
            col_all[ii] = __shfl(col, ii+kk);
            val_all[ii] = __shfl(val, ii+kk);
            //sum        += val_all[ii]*__ldg(B_offset+col_all[ii]);
            vals[   ii] = val_all[ii]*__ldg(B_offset+col_all[ii]);
            //vals[   ii] = __ldg(B_offset+col_all[ii]);
          }

          //if( warp_id==0 && blockIdx.y==0 )
          //  printf("row:%d,tid:%d,col_all:%d,ii:%d,load_id:%d,val:%f\n",row,thread_id,col_all>>6, ii, col_all+lane_id+((blockIdx.y&1)<<5), vals[ii]);

          #pragma unroll
          for( int ii=0; ii<TB; ii++ )
          {
            //val_all[ii] = __shfl(val, ii+kk);
            //sum += val_all[ii]*vals[ii];
            sum += vals[ii];
          //  if( threadIdx.x==1 && warp_id==0 && blockIdx.y==0 ) printf("tid:%d,ii:%d,val:%f\n", threadIdx.x, ii, vals[ii]);
          }

          //if( warp_id==0 && blockIdx.y==0 ) printf("tid:%d,val:%f\n", threadIdx.x, vals[0]);
        }
      }

      C_denseVal[C_offset] = sum;
    }
  } // spmmRowKernel2*/

  // Varies by B_ncols
  template<typename c, int TB>
  __global__ void spmmRowKernel3( const Index A_nrows, 
      const Index B_ncols, const Index A_ncols, const Index A_nvals,
      const Index* A_csrRowPtr, const Index* A_csrColInd, const c* A_csrVal, 
      const c* B_denseVal, c* C_denseVal )
  {
    float vals[TB];
    int   col_all[TB];
    float val_all[TB];

    int thread_id = blockDim.x*blockIdx.x+threadIdx.x; // global thrd idx
    int warp_id   = thread_id>>5;                      // global warp idx
    int lane_id   = thread_id & (32 - 1);
    int row       = warp_id;
    const c* B_offset = B_denseVal+lane_id+(blockIdx.y<<5);
    //int C_offset  = (row*B_ncols)+lane_id+(blockIdx.y<<5);
    int C_offset  = (lane_id+(blockIdx.y<<5))*A_nrows+row;

    //if( threadIdx.x==0 )
    //  printf("row:%d\n", row);

    if( row < A_nrows )
    {
      int row_start = __ldg(A_csrRowPtr+row);
      int row_end   = __ldg(A_csrRowPtr+row+1);

      int   col = -1;
      float val = 0.f;
      float sum = 0.f;
      int   jj  = row_start+lane_id;

      //TODO: add popc() and ballot to query which to shfl
      if( blockIdx.y!=gridDim.y-1 )
      {
        for( int jj_start=row_start; jj_start<row_end; jj_start+=32 )
        {
          //#pragma unroll
          //for( int ii=0; ii<TB; ii++ )
          //  vals[ii] = 0.f;
          if( jj<row_end )
          {
            col = __ldg(A_csrColInd+jj)*B_ncols;
            val = __ldg(A_csrVal+jj);
          }
          else
          {
            col = 0;
            val = 0.f;
          }
          jj+=32;
          //if( warp_id==0 ) printf("tid:%d,col:%d,val:%f\n", threadIdx.x, col, val);
          for( int kk=0; kk<32; kk+=TB )
          {
            #pragma unroll
            for( int ii=0; ii<TB; ii++ )
            {
              col_all[ii] = __shfl(col, ii+kk);
              val_all[ii] = __shfl(val, ii+kk);
              //sum        += val_all[ii]*__ldg(B_offset+col_all[ii]);
              vals[   ii] = val_all[ii]*__ldg(B_offset+col_all[ii]);
              //vals[   ii] = __ldg(B_offset+col_all[ii]);
            }

            //if( warp_id==0 && blockIdx.y==0 )
            //  printf("row:%d,tid:%d,col_all:%d,ii:%d,load_id:%d,val:%f\n",row,thread_id,col_all>>6, ii, col_all+lane_id+((blockIdx.y&1)<<5), vals[ii]);

            #pragma unroll
            for( int ii=0; ii<TB; ii++ )
            {
              //val_all[ii] = __shfl(val, ii+kk);
              //sum += val_all[ii]*vals[ii];
              sum += vals[ii];
            //  if( threadIdx.x==1 && warp_id==0 && blockIdx.y==0 ) printf("tid:%d,ii:%d,val:%f\n", threadIdx.x, ii, vals[ii]);
            }
            //if( warp_id==0 && blockIdx.y==0 ) printf("tid:%d,val:%f\n", threadIdx.x, vals[0]);
          }
        }
        C_denseVal[C_offset] = sum;
      }
      else
      {
        int leftover = B_ncols - (blockIdx.y<<5);
        for( int jj_start=row_start; jj_start<row_end; jj_start+=32 )
        {
          //#pragma unroll
          //for( int ii=0; ii<TB; ii++ )
          //  vals[ii] = 0.f;
          if( jj<row_end )
          {
            col = __ldg(A_csrColInd+jj)*B_ncols;
            val = __ldg(A_csrVal+jj);
          }
          else
          {
            col = 0;
            val = 0.f;
          }
          jj+=32;
          //if( jj_start<row_start+32*5 && warp_id==0 ) printf("tid:%d,col:%d,val:%f\n", threadIdx.x, col, val);
          for( int kk=0; kk<32; kk+=TB )
          {
              #pragma unroll
              for( int ii=0; ii<TB; ii++ )
              {
                col_all[ii] = __shfl(col, ii+kk);
                val_all[ii] = __shfl(val, ii+kk);
                //sum        += val_all[ii]*__ldg(B_offset+col_all[ii]);
                if( lane_id<leftover )
                  vals[ii]  = val_all[ii]*__ldg(B_offset+col_all[ii]);
                else
                  vals[ii]  = 0.f;
                //vals[   ii] = __ldg(B_offset+col_all[ii]);
                //if( jj_start<row_start+32*5 && thread_id<2 && warp_id==0 && blockIdx.y==0 )
                  //printf("row:%d,tid:%d,ii:%d,val:%f\n",row,thread_id, ii, vals[ii]);
              }

              #pragma unroll
              for( int ii=0; ii<TB; ii++ )
              {
                //val_all[ii] = __shfl(val, ii+kk);
                //sum += val_all[ii]*vals[ii];
                sum += vals[ii];
              //  if( threadIdx.x==1 && warp_id==0 && blockIdx.y==0 ) printf("tid:%d,ii:%d,val:%f\n", threadIdx.x, ii, vals[ii]);
              }
              //if( jj_start<row_start+32*5 && warp_id==0 && blockIdx.y==0 ) printf("str tid:%d,val:%f\n", threadIdx.x, sum);
              //if( jj_start>row_end-32*5 && warp_id==0 && blockIdx.y==0 ) printf("end tid:%d,val:%f\n", threadIdx.x, sum);
          }
        }
        if( lane_id<leftover )
          C_denseVal[C_offset] = sum;
      }
    }
  } // spmmRowKernel3

}  // backend
}  // graphblas

#endif  // GRB_BACKEND_APSPIE_KERNELS_SPMM_CU
